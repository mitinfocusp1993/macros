
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>

int convertTime(std::string str);
std::string timeToStr(int time);

int main() {
	std::string artfname;
	std::string sensefname;
	std::string outfname;
	int offset = 0;
	std::vector<size_t> intCols;
	std::vector<std::string> batchInfo;
	size_t artColumn = 1;
	bool batchInput = false;
	std::string batchfname;
	std::ifstream batchfile;
	std::string batchline;
	std::ofstream intfile;

	std::cout << "Enter batch filename (blank allows manual input): ";
	std::getline(std::cin, batchfname);
	if (batchfname != "") batchInput = true;
	std::cout << std::endl;

	if (!batchInput) {
		std::cout << "Enter art-line file name: ";
		std::getline(std::cin, artfname);
		std::cout << std::endl;

		std::cout << "Enter sensor file name: ";
		std::getline(std::cin, sensefname);
		std::cout << std::endl;

		std::cout << "Enter output file name: ";
		std::getline(std::cin, outfname);
		std::cout << std::endl;

		std::cout << "Enter sensor columns to put in interval file: ";
		std::string colstr;
		std::getline(std::cin, colstr);
		std::cout << std::endl;

		if (colstr != "") {
			intCols.clear();
			std::stringstream colss(colstr);
			size_t col;
			while (colss >> col) {
				intCols.push_back(col);
			}
		}

		std::cout << "Enter art-line column to use in interval file: ";
		std::cin >> artColumn;
		std::cout << std::endl;

		std::cout << "Enter offset (artline - sensor times in seconds): ";
		std::cin >> offset;
		std::cout << std::endl;
	}
	else {
		batchfile.open(batchfname);
		std::getline(batchfile, batchline);
		std::stringstream intss;
		intss << "int_" << batchfname;
		intfile.open(intss.str());
	}

	size_t numFiles = 0;
	do {
		numFiles++;
		std::cout << "Starting file #" << numFiles << std::endl;
		if (batchInput) {
			std::stringstream batchss;
			batchss << batchline;
			batchss >> artfname >> sensefname >> outfname >> offset >> artColumn;
			size_t scol;
			intCols.clear();
			for (size_t i = 0; i<4 && (batchss >> scol); i++) 
				intCols.push_back(scol);
			std::string bInfo;
			batchInfo.clear();
			while (batchss >> bInfo)
				batchInfo.push_back(bInfo);
		}
		std::ifstream artfile(artfname);
		std::ifstream sensefile(sensefname);
		std::ofstream outfile(outfname);

		if (!batchInput) {
			std::stringstream intss;
			intss << "int_" << outfname;
			intfile.open(intss.str());
		}

		std::string dum;
		getline(artfile, dum);	//remove header
		std::string artline;
		if (!getline(artfile, artline)) {
			std::cout << "Artfile empty" << std::endl;
			return 0;
		}

		size_t slinenum = 0;
		size_t curIntStart = 1;
		bool intOpen = false;
		std::string senseline;
		float artRes = 0;
		std::vector<std::string> artcols;
		size_t numLinesOutput = 0;
		size_t outfileNumber = 1;
		size_t numUnmatchedLines = 0;
		while (getline(sensefile, senseline)) {
			slinenum++;
			std::stringstream senselss;
			senselss << senseline;
			std::vector<std::string> sensecolumns;
			while (senselss >> dum)
				sensecolumns.push_back(dum);
			int sensetime = convertTime(sensecolumns.back()) + offset;

			bool foundArtLine = false;
			do  {
				std::stringstream artss;
				artss << artline;
				std::string arttimestr;
				getline(artss, arttimestr, ',');
				int arttime = convertTime(arttimestr);

				if (abs(arttime - sensetime) <= 30) {
					std::string column;
					artcols.clear();
					size_t col = 1;
					while (getline(artss, column, ',')) {
						col++;
						artcols.push_back(column);
						if (col == artColumn) {
							std::stringstream colss;
							colss << column;
							colss >> artRes;
						}
					}
					outfile << artRes;
					for (size_t i = 0; i < sensecolumns.size(); i++) {
						bool colSelected = false;
						if (i >= sensecolumns.size() - 2)
							colSelected = true;
						else {
							for (size_t j = 0; j < intCols.size(); j++) {
								if (intCols[j] == i + 1) {
									colSelected = true;
									break;
								}
							}
						}
						if (colSelected)
							outfile << "," << sensecolumns[i];
					}
					for (size_t i = 0; i < batchInfo.size(); i++) {
						outfile << "," << batchInfo[i];
					}
					for (size_t i = 0; i < artcols.size(); i++) {
						outfile << "," << artcols[i];
					}
					outfile << std::endl;
					numLinesOutput++;
					if (numLinesOutput >= 1000000) {
						numLinesOutput = 0;
						outfileNumber++;
						std::stringstream outnewss;
						outnewss << outfname << "-" << outfileNumber;
						outfile.close();
						outfile.open(outnewss.str());
					}
					foundArtLine = true;
					if (!intOpen) {
						curIntStart = slinenum;
						intOpen = true;
					}
					break;
				}
				else if (arttime - sensetime > 30) {
					//std::cout << "Failed to find art-line match for sensor data at " << timeToStr(sensetime) << std::endl;
					foundArtLine = true;
					break;
				}
				if (intOpen) {
					for (size_t i = 0; i < intCols.size(); i++) {
						intfile << sensefname << " " << intCols[i] << " " << curIntStart << " " << slinenum - 1 << " " << artRes << " ";
						for (size_t i = 0; i < batchInfo.size(); i++) {
							intfile << batchInfo[i] << " ";
						}
						for (size_t i = 0; i < artcols.size(); i++) {
							if (artcols[i] == "")
								intfile << 0;
							intfile << artcols[i] << " ";
						}
						intfile << std::endl;
					}
					intOpen = false;
				}
				artRes = 0;
			} while (getline(artfile, artline));

			if (foundArtLine == false) {
				numUnmatchedLines++;
				if (numUnmatchedLines >= 5) {
					//std::cout << "Artline ended without match for " << timeToStr(sensetime) << std::endl;
					break;
				}
				artfile.clear();
				artfile.seekg(0, std::ios::beg);
			}
			else
				numUnmatchedLines = 0;
		}
	} while (batchInput && std::getline(batchfile, batchline));

	system("pause");
}

int convertTime(std::string str) {
	int hour = 0;
	int minute = 0;
	int second = 0;
	char dum;
	std::string pm;

	std::stringstream strs;
	strs << str;
	strs >> hour;
	strs >> dum;	//":"
	strs >> minute;
	strs >> dum;	//":"
	strs >> second;
	strs >> pm;

	int time = 3600 * hour + 60 * minute + second;
	if (pm == "PM")
		time += 12 * 3600;

	return time;
}
std::string timeToStr(int time) {
	if (time < 0)
		return "0";
	std::stringstream ss;
	ss << time / 3600 << ":";
	time = time - 3600 * (time / 3600);
	ss << time / 60 << ":";
	time = time - 60 * (time / 60);
	ss << time;
	return ss.str();
}