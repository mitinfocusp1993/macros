
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <list>

#define datastring ""
//#define datastring "D:/trivialNetworkTest/sethsensor/"

struct SensorConvDatasetInfo {
	std::ifstream* datafile;
	size_t windowSize;
	size_t firstQualityColumn;
	float qualityThresh;
	size_t firstDataColumn;
	size_t lastDataColumn;
	std::string outputName;
	size_t outNum = 0;
};

struct IntervalData {
	std::string fname;
	size_t startIndex;
	size_t endIndex;
	float output;
	SensorConvDatasetInfo* info;
};

void convertInterval(IntervalData* intData);
void convertTrainset(SensorConvDatasetInfo* info);

int main() {
	SensorConvDatasetInfo dataInfo;

	std::string datafname;
	std::cout << "Enter data file: ";
	std::cin >> datafname;

	std::ifstream datafile(datastring + datafname);
	if (!datafile.is_open()) {
		std::cout << "Can't find datafile" << std::endl;
		system("pause");
		return 0;
	}
	dataInfo.datafile = &datafile;

	std::cout << "Enter window size: ";
	std::cin >> dataInfo.windowSize;

	std::cout << "Enter first data column (starting at 0): ";
	std::cin >> dataInfo.firstDataColumn;
	
	std::cout << "Enter last data column: ";
	std::cin >> dataInfo.lastDataColumn;

	std::cout << "Enter quality threshold: ";
	std::cin >> dataInfo.qualityThresh;

	std::cout << "Enter first quality column: ";
	std::cin >> dataInfo.firstQualityColumn;

	std::cout << "Enter output file name: ";
	std::cin >> dataInfo.outputName;

	convertTrainset(&dataInfo);

	system("pause");
}

void convertTrainset(SensorConvDatasetInfo* info) {
	std::string line;

	while (std::getline((*info->datafile), line)) {
		IntervalData intData;
		(std::stringstream(line)) >> intData.fname >> intData.startIndex >> intData.endIndex >> intData.output;
		intData.info = info;

		std::cout << "Reading interval: " << intData.fname << " " << intData.startIndex << " " << intData.endIndex << " " << intData.output << std::endl;
		info->outNum++;

		convertInterval(&intData);
	}
}

void convertInterval(IntervalData* intData) {
	std::ifstream infile(datastring + intData->fname);
	if (!infile.is_open()) {
		std::cout << "Couldn't open file " << datastring + intData->fname << std::endl;
		return;
	}
	std::stringstream outss;
	outss << datastring << intData->info->outputName << "_" << intData->info->outNum;
	FILE* outfile = fopen(outss.str().c_str(), "wb");
	float header = 0;
	fwrite(&header, sizeof(float), 1, outfile);
	fwrite(&header, sizeof(float), 1, outfile);

	std::string line;
	for (size_t i = 0; i < intData->startIndex; i++) {
		if (infile.ignore(10000, infile.widen('\n'))){
			//just skipping the line
		}
		else {
			std::cout << "Some sort of error skipping initial lines of file " << std::endl;
			system("pause");
		}
	}

	std::vector<std::list<long long>> data;
	data.resize(intData->info->lastDataColumn - intData->info->firstDataColumn + 1);
	std::vector<float> quality;
	quality.resize(data.size());

	for (size_t i = intData->startIndex; i < intData->endIndex; i++) {
		std::getline(infile, line);
		std::stringstream lss(line);

		std::string dum;
		quality.clear();
		for (size_t c = 0; std::getline(lss, dum, ','); c++) {
			if (c >= intData->info->firstDataColumn && c <= intData->info->lastDataColumn) {
				long long val;
				(std::stringstream(dum)) >> val;
				size_t colNum = c - intData->info->firstDataColumn;
				data[colNum].push_back(val);
				if (data[colNum].size() > intData->info->windowSize)
					data[colNum].pop_front();
			}
			else if (c >= intData->info->firstQualityColumn) {
				float val;
				(std::stringstream(dum)) >> val;
				if (c - intData->info->firstQualityColumn < data.size())
					quality.push_back(val);
			}
		}

		for (size_t q = 0; q < quality.size(); q++) {
			if (fabs(quality[q]) >= intData->info->qualityThresh && data[q].size() == intData->info->windowSize) {
				for (size_t invert = 0; invert < 2; invert++) {
					std::vector<float> inputs;
					long long initVal = (invert == 0 ? data[q].front() : -data[q].front());
					long long maxVal = initVal;
					long long minVal = initVal;
					for (std::list<long long>::iterator it = data[q].begin(); it != data[q].end(); it++) {
						long long val;
						if (invert == 0)
							val = (*it);
						else
							val = -(*it);
						maxVal = std::max(maxVal, val);
						minVal = std::min(minVal, val);
					}

					for (std::list<long long>::iterator it = data[q].begin(); it != data[q].end(); it++) {
						long long val;
						if (invert == 0)
							val = (*it);
						else
							val = -(*it);

						long long valDiff = val - minVal;
						long long maxDiff = maxVal - minVal;

						if (maxDiff > 0)
							inputs.push_back(2.0f*valDiff / maxDiff - 1.0f);
						else
							inputs.push_back(0.0f);
					}

					fwrite(&intData->output, sizeof(float), 1, outfile);
					fwrite(&quality[q], sizeof(float), 1, outfile);
					if (inputs.size() != intData->info->windowSize)
						std::cout << "Invalid size window!" << std::endl;
					else
						fwrite(&inputs[0], sizeof(float), inputs.size(), outfile);
				}
			}
		}
	}
}