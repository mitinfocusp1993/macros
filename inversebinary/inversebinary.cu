
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>

#define datastring ""
//#define datastring "D:/trivialNetworkTest/sethsensor/"

#define NUM_INPUTS 512
#define NUM_OUTPUTS 2//5
#define NUM_SILENT_OUTPUTS 0//12
#define HEADER_SIZE 8

int main() {
	std::string filefname;
	std::cout << "Enter file list: ";
	std::cin >> filefname;

	std::ifstream filelist((std::string)datastring + filefname);
	std::string line;
	while (std::getline(filelist, line)) {
		std::string fname;
		std::stringstream(line) >> fname;
		FILE* infile = fopen(fname.c_str(), "rb");
		FILE* outfile = fopen(("inv_" + fname).c_str(), "wb");

		char* buffer[HEADER_SIZE];
		fread(&buffer, HEADER_SIZE, 1, infile);
		fwrite(&buffer, HEADER_SIZE, 1, outfile);

		size_t numColumns = NUM_INPUTS + NUM_OUTPUTS + NUM_SILENT_OUTPUTS;
		std::vector<float> columns(numColumns);
		while (fread(&columns[0], sizeof(float), numColumns, infile) == numColumns) {
			fwrite(&columns[0], sizeof(float), numColumns, outfile);
			//flip outputs but not silent outputs
			for (size_t c = 0; c < NUM_OUTPUTS; c++)
				columns[c] = -columns[c];
			for (size_t c = NUM_OUTPUTS + NUM_SILENT_OUTPUTS; c < numColumns; c++)
				columns[c] = -columns[c];
			fwrite(&columns[0], sizeof(float), numColumns, outfile);
		}
	}
}
