
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include <vector>
#define _USE_MATH_DEFINES
#include <math.h>
#include <time.h>

#define imagestring "D:/stopsearch/"
#define datastring "D:/stopsearch/data/"
#define dataname "toydata_hawaii"
#define imagename "hawaiiconv"

#define NUM_EVENTS 1e6
#define NUM_VARS 2


void throwError(std::string err) {
	std::cout << err << std::endl;
	system("pause");
	throw std::runtime_error(err);
}

bool evaluateEvent(std::vector<float>* inputs, std::vector<std::vector<float>>* goalimage) {
	if (inputs->size() != NUM_VARS)
		throwError("Invalid input size");

	size_t bin0 = (size_t)((*inputs)[0] * goalimage->size());
	size_t bin1 = (size_t)((*inputs)[1] * (*goalimage)[bin0].size());
		
	return (*goalimage)[bin0][bin1] > 0;
}

int main() {
	srand((size_t)time(NULL));
	std::stringstream outss;
	outss << datastring << dataname;
	std::ofstream signalfile(outss.str() + "_signal");
	std::ofstream backgroundfile(outss.str() + "_background");

	std::stringstream imagess;
	imagess << imagestring << imagename;
	std::ifstream imagestream(imagess.str());

	std::string line;
	std::vector<std::vector<float>> goalimage;
	while (std::getline(imagestream, line)) {
		std::vector<float> iline;
		std::stringstream lss(line);
		float val;
		while (lss >> val) {
			iline.push_back(val);
		}
		goalimage.push_back(iline);
	}

	for (size_t i = 0; i < NUM_EVENTS; i++) {
		std::vector<float> inputs(NUM_VARS);
		for (size_t j = 0; j < NUM_VARS; j++) {
			inputs[j] = 1.0f*(rand() % 10000) / 10000;
		}

		std::ofstream* outfile = (evaluateEvent(&inputs, &goalimage) ? &signalfile : &backgroundfile);
		for (size_t j = 0; j < inputs.size(); j++) {
			(*outfile) << inputs[j] << " ";
		}
		(*outfile) << std::endl;
	}
}