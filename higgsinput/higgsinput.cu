
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include <vector>
#define _USE_MATH_DEFINES
#include <math.h>

#define datastring "D:/higgs/data/"

std::vector<float> convertFeatures(std::vector<float> inputs);

int main() {
	std::string infname;
	std::string outfname;
	bool convertInputs = false;
	bool trainingFile = false;
	std::cout << "Enter input file name: ";
	std::cin >> infname;
	std::cout << "Enter output file prefix: ";
	std::cin >> outfname;
	std::cout << "Convert inputs? ";
	std::cin >> convertInputs;
	std::cout << "Does this file have training information? ";
	std::cin >> trainingFile;

	std::ifstream infile(datastring + infname);
	std::ofstream signalfile(datastring + outfname + "_s");
	std::ofstream backgroundfile(datastring + outfname + "_b");
	std::ofstream totaloutfile(datastring + outfname);

	if (!infile.is_open()) {
		std::cout << "Couldn't find input file." << std::endl;
		system("pause");
		return 0;
	}

	std::string line;
	std::getline(infile, line);	//header

	while (std::getline(infile, line)) {
		std::string tok;
		std::stringstream lss(line);
		size_t id;
		float weight = 1.0f;
		std::string type;

		std::getline(lss, tok, ',');
		std::stringstream tss(tok);
		tss >> id;

		std::vector<float> features(30);
		for (size_t i = 0; i < 30; i++) {
			std::getline(lss, tok, ',');
			tss.clear();
			tss.str(tok);
			tss >> features[i];
		}

		//output
		std::ofstream* outfile;

		if (trainingFile) {
			std::getline(lss, tok, ',');
			tss.clear();
			tss.str(tok);
			tss >> weight;

			std::getline(lss, tok, ',');
			tss.clear();
			tss.str(tok);
			tss >> type;

			if (type == "s")
				outfile = &signalfile;
			else
				outfile = &backgroundfile;
		}
		else {
			outfile = &totaloutfile;
		}

		if (convertInputs)
			features = convertFeatures(features);
		(*outfile) << weight << " ";
		for (size_t i = 0; i < features.size(); i++) {
			(*outfile) << features[i] << " ";
		}
		(*outfile) << std::endl;
	}
}

std::vector<float> convertFeatures(std::vector<float> inputs) {
	std::vector<float> convFeatures;

	float val;
	//estimated higgs mass
	val = inputs[0];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//transverse mass between missing transverse energy and lepton
	val = inputs[1];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//visible invariant mass
	val = inputs[2];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//modulus of p_t of tau, lepton, and miss_pt
	val = inputs[3];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//jet-jet d_eta
	val = inputs[4];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);

	//jet-jet invariant mass
	val = inputs[5];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//jet-jet eta product
	val = inputs[6];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);

	//hadron tau-lepton R separation
	val = inputs[7];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);

	//total p_t modulus (without additional jets)
	val = inputs[8];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//total p_t modulus
	val = inputs[9];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//lep-tau ratio
	val = inputs[10];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//m_et-phi centrality
	val = inputs[11];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);

	//lep-eta centrality
	val = inputs[12];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);

	//tau-pt
	val = inputs[13];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//tau-eta
	val = inputs[14];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);

	//tau-phi
	/*
	val = inputs[15];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);
		*/

	//lep-pt
	val = inputs[16];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//lep-eta
	val = inputs[17];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);

	//lep-phi
	/*
	val = inputs[18];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);
		*/

	//met
	val = inputs[19];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//met-phi
	/*
	val = inputs[20];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);
		*/

	//met-sum_et
	val = inputs[21];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//jet num
	val = inputs[22];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);

	//jet leading pt
	val = inputs[23];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//jet leading eta
	val = inputs[24];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);

	//jet leading phi
	/*
	val = inputs[25];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);
		*/

	//jet subleading pt
	val = inputs[26];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//jet subleading eta
	val = inputs[27];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//jet subleading phi
	/*
	val = inputs[28];
	if (val != -999)
		convFeatures.push_back(val);
	else
		convFeatures.push_back(NAN);
		*/

	//jet sum pt
	val = inputs[29];
	if (val != -999)
		convFeatures.push_back(std::log(val));
	else
		convFeatures.push_back(NAN);

	//Additional features

	//tau: 15, lep: 18, met: 20, jet1: 25, jet2: 28
	//tau-lep phi diff
	float phi1 = inputs[15];
	float phi2 = inputs[18];
	float convPhi;
	if (phi1 == -999 || phi2 == -999)
		convPhi = NAN;
	else {
		convPhi = fabs(phi1 - phi2);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
	}
	convFeatures.push_back(convPhi);

	//tau-met phi diff
	phi1 = inputs[15];
	phi2 = inputs[20];
	if (phi1 == -999 || phi2 == -999)
		convPhi = NAN;
	else {
		convPhi = fabs(phi1 - phi2);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
	}
	convFeatures.push_back(convPhi);

	//tau-jet1 phi diff
	phi1 = inputs[15];
	phi2 = inputs[25];
	if (phi1 == -999 || phi2 == -999)
		convPhi = NAN;
	else {
		convPhi = fabs(phi1 - phi2);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
	}
	convFeatures.push_back(convPhi);

	//tau-jet2 phi diff
	phi1 = inputs[15];
	phi2 = inputs[28];
	if (phi1 == -999 || phi2 == -999)
		convPhi = NAN;
	else {
		convPhi = fabs(phi1 - phi2);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
	}
	convFeatures.push_back(convPhi);

	//lep-met phi diff
	phi1 = inputs[18];
	phi2 = inputs[20];
	if (phi1 == -999 || phi2 == -999)
		convPhi = NAN;
	else {
		convPhi = fabs(phi1 - phi2);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
	}
	convFeatures.push_back(convPhi);

	//lep-jet1 phi diff
	phi1 = inputs[18];
	phi2 = inputs[25];
	if (phi1 == -999 || phi2 == -999)
		convPhi = NAN;
	else {
		convPhi = fabs(phi1 - phi2);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
	}
	convFeatures.push_back(convPhi);

	//lep-jet2 phi diff
	phi1 = inputs[18];
	phi2 = inputs[28];
	if (phi1 == -999 || phi2 == -999)
		convPhi = NAN;
	else {
		convPhi = fabs(phi1 - phi2);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
	}
	convFeatures.push_back(convPhi);

	//met-jet1 phi diff
	phi1 = inputs[20];
	phi2 = inputs[25];
	if (phi1 == -999 || phi2 == -999)
		convPhi = NAN;
	else {
		convPhi = fabs(phi1 - phi2);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
	}
	convFeatures.push_back(convPhi);

	//met-jet2 phi diff
	phi1 = inputs[20];
	phi2 = inputs[28];
	if (phi1 == -999 || phi2 == -999)
		convPhi = NAN;
	else {
		convPhi = fabs(phi1 - phi2);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
	}
	convFeatures.push_back(convPhi);

	//jet1-jet2 phi diff
	phi1 = inputs[25];
	phi2 = inputs[28];
	if (phi1 == -999 || phi2 == -999)
		convPhi = NAN;
	else {
		convPhi = fabs(phi1 - phi2);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
	}
	convFeatures.push_back(convPhi);

	return convFeatures;
}