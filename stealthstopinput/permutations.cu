#include "hip/hip_runtime.h"
#include "permutations.cuh"

PermutationArray createPermutation(size_t numIndices) {
	PermutationArray perm;
	perm.indices.resize(numIndices);
	perm.directions.resize(numIndices);
	for (size_t i = 0; i < numIndices; i++) {
		perm.indices[i] = i + 1;
		if (i == 0)
			perm.directions[i] = 0;
		else
			perm.directions[i] = -1;
	}
	return perm;
}

bool iteratePermutation(PermutationArray* perm) {
	size_t maxMover = 0;
	size_t maxIndex = 0;	//index of index array :) confusing
	for (size_t i = 0; i < perm->indices.size(); i++) {
		if (perm->indices[i] > maxMover && perm->directions[i] != 0) {
			maxMover = perm->indices[i];
			maxIndex = i;
		}
	}
	if (maxMover == 0)
		return false;

	//swap max index in chosen direction
	if ((size_t)(maxIndex + perm->directions[maxIndex] >= perm->indices.size())) {
		std::cout << "Tried to swap permutation index out of bounds" << std::endl;
		throw new std::runtime_error("Tried to swap permutation index out of bounds");
	}

	size_t tmpVal = perm->indices[maxIndex];
	size_t tmpDir = perm->directions[maxIndex];
	size_t swapIndex = maxIndex + perm->directions[maxIndex];
	perm->indices[maxIndex] = perm->indices[swapIndex];
	perm->indices[swapIndex] = tmpVal;
	perm->directions[maxIndex] = perm->directions[swapIndex];
	perm->directions[swapIndex] = tmpDir;

	//set direction of swapped index to zero if it runs into the edge of the array or a larger index
	if (swapIndex == 0 || swapIndex == perm->indices.size() - 1 || perm->indices[swapIndex + perm->directions[swapIndex]] > perm->indices[swapIndex])
		perm->directions[swapIndex] = 0;

	//set all indices greater than the swapped index to move towards the swapped index
	for (size_t i = 0; i < perm->indices.size(); i++) {
		if (perm->indices[i] > perm->indices[swapIndex]) {
			if (i < swapIndex)
				perm->directions[i] = 1;
			else
				perm->directions[i] = -1;
		}
	}

	return true;
}