#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include "trivialdampingfilterFourier.cuh"
#include "trivialdampingfilterInterpolate.cuh"
#include <chrono>

#define datastring ""
//#define datastring "D:/trivialNetworkTest/peakbinpacker/"

#define NUM_INPUTS 512
#define NUM_OUTPUTS 4
#define NUM_SILENT_OUTPUTS 12
#define HEADER_SIZE 8

#define MATH_PI 3.14159265358979323846

#define DERIV_SMOOTHING_RANGE 3

#define REMOVE_FLATLINES
#define FLATLINE_SEQ_POINTS 7

#define PRIMARY_OUTPUT_INDEX 1

#define FOURIER_PATCH_WIDTH_POW 9
#define FOURIER_PATCH_WIDTH (pow(2,FOURIER_PATCH_WIDTH_POW))
#define FOURIER_PATCH_PEAK_EDGE_SHRINK 0.3
#define FOURIER_PATCH_VALLEY_EDGE_SHRINK 0.05
#define FOURIER_SCALE_FACTOR 60.0f

#define NUM_SAMPLES_PER_PERSON 5000

std::vector<float> fourierRe;
std::vector<float> fourierIm;
std::vector<float> fourierAvg;
std::vector<float> fourierPatch;
std::vector<float> fourierFit;
std::vector<float> fourierFitCoef;
float fourierFitMinAmp = 2.0f;
float fourierFitMinMean = 40.0f;
float fourierFitMaxMean = 1000.0f;
float fourierFitMaxSigma = 100.0f;
float fourierFitMinMagRatio = 0.7f;
float fourierFitMinWidthRatio = 0.1f;

std::chrono::system_clock::time_point markedStartTime;

template <typename T> void randomizeVector(std::vector<T>* vec) {
	for (size_t i = 0; i < vec->size(); i++) {
		size_t j = (RAND_MAX*rand() + rand()) % vec->size();
		T tmp = (*vec)[i];
		(*vec)[i] = (*vec)[j];
		(*vec)[j] = tmp;
	}
}

void markTime() {
	markedStartTime = std::chrono::high_resolution_clock::now();
}

long long getTimeSinceMark() {
	auto elapsed = std::chrono::high_resolution_clock::now() - markedStartTime;
	long long time = std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
	return time / 1000000;
}

int main() {
	srand((size_t)time(NULL));

	std::string filelist = "filelist";
	std::ifstream infilelist(datastring + filelist);
	if (!infilelist.is_open()) {
		std::cout << "Couldn't open file list " << datastring << filelist << std::endl;
	}

	std::string dampfolder;
	std::cout << "Enter folder to store damped results: ";
	std::cin >> dampfolder;

	if (dampfolder == "") {
		std::cout << "Must enter non-empty dampfolder" << std::endl;
		system("pause");
		return;
	}

	std::string fileline;
	std::vector<float> waveform(NUM_INPUTS);
	std::vector<float> deriv1(NUM_INPUTS);
	std::vector<float> deriv2(NUM_INPUTS);
	std::vector<float> peaks(NUM_INPUTS);
	while (std::getline(infilelist, fileline)) {
		std::string fname;
		(std::stringstream(fileline)) >> fname;
		std::cout << "Reading file " << fname << ": ";
		markTime();

		char* buf[HEADER_SIZE];
		FILE* infile = fopen((datastring + fname).c_str(), "rb");
		_fseeki64(infile, 0, SEEK_SET);
		fread(&buf, HEADER_SIZE, 1, infile);

		FILE* outfile = fopen((datastring + dampfolder + "/" + fname).c_str(), "wb");
		fwrite(&buf, HEADER_SIZE, 1, outfile);

		size_t numColumns = 2*NUM_INPUTS + NUM_OUTPUTS + NUM_SILENT_OUTPUTS;	//Peak data must be included.

		std::vector<float> columns(numColumns);
		std::vector<size_t> sampleIndices;
		float avgOut = 0;
		while (fread(&columns[0], sizeof(float), numColumns, infile) == numColumns) {
			sampleIndices.push_back(sampleIndices.size());
			avgOut += columns[PRIMARY_OUTPUT_INDEX];
		}
		randomizeVector(&sampleIndices);
		avgOut /= sampleIndices.size();
		std::cout << "Average Output: " << avgOut << " ";

		size_t curSampleIndex = 0;

		size_t numUnderdampedEntries = 0;
		size_t numFlatEntries = 0;
		_fseeki64(infile, HEADER_SIZE, SEEK_SET);
		for (size_t ent = 0; ent < NUM_SAMPLES_PER_PERSON; ent++) {
			size_t curSample = sampleIndices[curSampleIndex];
			_fseeki64(infile, HEADER_SIZE + numColumns*curSample*sizeof(float), SEEK_SET);
			fread(&columns[0], numColumns, sizeof(float), infile);

			float curIndexFloat = (float)curSample;
			curSampleIndex++;
			if (curSampleIndex > sampleIndices.size()) {
				curSampleIndex = 0;
				randomizeVector(&sampleIndices);
			}

#ifdef REMOVE_FLATLINES
			float lastVal = 9999;
			size_t flatSize = 0;
			for (size_t in = 0; in < NUM_INPUTS; in++) {
				float val = columns[in + NUM_OUTPUTS + NUM_SILENT_OUTPUTS];
				if (val == lastVal)
					flatSize++;
				else {
					flatSize = 0;
					lastVal = val;
				}
				if (flatSize > FLATLINE_SEQ_POINTS)
					break;
			}
			if (flatSize > FLATLINE_SEQ_POINTS) {
				numFlatEntries++;
				ent--;
				continue;
			}
#endif
			//extract waveform and derivatives
			float minVal = 99999;
			float maxVal = -99999;
			for (size_t i = 0; i < NUM_INPUTS; i++) {
				waveform[i] = columns[NUM_OUTPUTS + NUM_SILENT_OUTPUTS + i];
				minVal = std::min(minVal, waveform[i]);
				maxVal = std::max(maxVal, waveform[i]);
				peaks[i] = columns[NUM_OUTPUTS + NUM_SILENT_OUTPUTS + NUM_INPUTS + i];
			}

			deriv1.clear();
			for (size_t i = 0; i < DERIV_SMOOTHING_RANGE; i++)
				deriv1.push_back(0);
			for (size_t i = 0; i < waveform.size() - 2 * DERIV_SMOOTHING_RANGE; i++) {
				float backAvg = 0;
				for (size_t j = 0; j < DERIV_SMOOTHING_RANGE; j++) {
					backAvg += waveform[i + j];
				}
				backAvg /= DERIV_SMOOTHING_RANGE;
				float forwardAvg = 0;
				for (size_t j = 0; j < DERIV_SMOOTHING_RANGE; j++) {
					forwardAvg += waveform[i + DERIV_SMOOTHING_RANGE + j + 1];
				}
				forwardAvg /= DERIV_SMOOTHING_RANGE;
				deriv1.push_back(forwardAvg - backAvg);
			}
			for (size_t i = 0; i < DERIV_SMOOTHING_RANGE; i++)
				deriv1.push_back(0);

			deriv2.clear();
			for (size_t i = 0; i < DERIV_SMOOTHING_RANGE; i++)
				deriv2.push_back(0);
			for (size_t i = 0; i < deriv1.size() - 2 * DERIV_SMOOTHING_RANGE; i++) {
				float backAvg = 0;
				for (size_t j = 0; j < DERIV_SMOOTHING_RANGE; j++) {
					backAvg += deriv1[i + j];
				}
				backAvg /= DERIV_SMOOTHING_RANGE;
				float forwardAvg = 0;
				for (size_t j = 0; j < DERIV_SMOOTHING_RANGE; j++) {
					forwardAvg += deriv1[i + DERIV_SMOOTHING_RANGE + j + 1];
				}
				forwardAvg /= DERIV_SMOOTHING_RANGE;
				deriv2.push_back(forwardAvg - backAvg);
			}
			for (size_t i = 0; i < DERIV_SMOOTHING_RANGE; i++)
				deriv2.push_back(0);

			//compute fourier transform
			fourierRe.clear();
			fourierRe.resize(FOURIER_PATCH_WIDTH);
			fourierIm.clear();
			fourierIm.resize(FOURIER_PATCH_WIDTH);
			fourierAvg.clear();
			fourierAvg.resize(FOURIER_PATCH_WIDTH);
			fourierPatch.clear();
			fourierPatch.resize(FOURIER_PATCH_WIDTH);
			fourierFit.clear();
			fourierFit.resize(FOURIER_PATCH_WIDTH / 2);

			std::vector<size_t> peakLocs;
			std::vector<size_t> valleyLocs;
			if (peaks.size() > 0) {
				for (size_t i = 0; i < peaks.size(); i++) {
					if (peaks[i] == 1)
						peakLocs.push_back(i);
					if (peaks[i] == -1)
						valleyLocs.push_back(i);
				}
			}

			size_t nextValley = 0;
			std::vector<float> waveformPatch;
			size_t numPatches = 0;
			for (size_t p = 0; p < peakLocs.size(); p++) {
				size_t patchStart = peakLocs[p];
				while (nextValley < valleyLocs.size() && valleyLocs[nextValley] < patchStart)
					nextValley++;

				if (nextValley >= valleyLocs.size())
					break;

				numPatches++;
				size_t patchEnd = valleyLocs[nextValley];
				size_t patchSize = patchEnd - patchStart;

				patchStart += patchSize*FOURIER_PATCH_PEAK_EDGE_SHRINK;
				patchEnd -= patchSize*FOURIER_PATCH_VALLEY_EDGE_SHRINK;
				patchSize = patchEnd - patchStart;

				waveformPatch.resize(patchSize);

				for (size_t i = 0; i < patchSize; i++) {
					waveformPatch[i] = deriv2[patchStart + i];
				}

				interpolate(&waveformPatch, &fourierRe, FOURIER_PATCH_WIDTH);

				FFT(1, FOURIER_PATCH_WIDTH_POW, &fourierRe[0], &fourierIm[0]);

				for (size_t i = 0; i < fourierRe.size(); i++) {
					fourierRe[i] = sqrt(fourierRe[i] * fourierRe[i] + fourierIm[i] * fourierIm[i]);
				}

				interpolate(&fourierRe, &fourierPatch, FOURIER_PATCH_WIDTH*FOURIER_PATCH_WIDTH / patchSize);
				for (size_t i = 0; i < fourierPatch.size(); i++) {
					fourierAvg[i] += fourierPatch[i];
				}
			}
			for (size_t i = 0; i < fourierAvg.size(); i++) {
				fourierAvg[i] /= numPatches;
			}

			for (size_t i = 0; i < fourierAvg.size(); i++) {
				if (maxVal > minVal)
					fourierAvg[i] = FOURIER_SCALE_FACTOR*fourierAvg[i] / (maxVal - minVal);// 2.0f*(fourierAvg[i] - minFour) / (maxFour - minFour) - 1.0f;
				else
					fourierAvg[i] = 0.0f;
			}

			fourierBWFit(&fourierAvg[0], &fourierFit[0], &fourierFitCoef, 128);// fourierAvg.size() / 2);

			float fourierFitMagRatio = (fourierFitCoef[8] > 0 ? fourierFitCoef[4] / fourierFitCoef[8] : 9999);
			float fourierFitWidthRatio = fourierFitCoef[4] / fourierFitCoef[3];
			if (fourierFitCoef[4] > fourierFitMinAmp && fourierFitCoef[2] > fourierFitMinMean && fourierFitCoef[2] < fourierFitMaxMean && fourierFitCoef[3] < fourierFitMaxSigma && fourierFitMagRatio > fourierFitMinMagRatio && fourierFitWidthRatio > fourierFitMinWidthRatio) {
				numUnderdampedEntries++;
				ent--;
				continue;
			}

			fwrite(&curIndexFloat, sizeof(float), 1, outfile);
			fwrite(&columns[0], sizeof(float), numColumns, outfile);
		}

		std::cout << "Done. (" << getTimeSinceMark() << " s) Total: " << sampleIndices.size() << " Flat: " << numFlatEntries << " Underdamped: " << numUnderdampedEntries << std::endl;
		fclose(outfile);
		fclose(infile);
	}

	system("pause");
	system("pause");
}
