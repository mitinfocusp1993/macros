#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include <vector>
#define _USE_MATH_DEFINES
#include <math.h>
#include "clusterdistStructDef.cuh"

#define datafname "D:/stopSearch/weights/deep_350-200_FixPhiWBMatchedNoMETEtaBTag_L1_2e-5/actMaxL5N10_45/raw_output"
#define outfolder "D:/stopSearch/weights/deep_350-200_FixPhiWBMatchedNoMETEtaBTag_L1_2e-5/actMaxClusterL5N10_45_lowpt/"

float displayCorrelationHistogram(Histogram2D hist, std::ofstream* resfile);

bool includeEvent(std::vector<float>* inputs) {
	//return (*inputs)[0] > 3	&& (*inputs)[0] < 5 && (*inputs)[1] > 3.2 && (*inputs)[1] < 4 && (*inputs)[5] > 3.2 && (*inputs)[5] < 4;
	return (*inputs)[11] > 1.5 && (*inputs)[11] < 2 && (*inputs)[13] > 0.25 && (*inputs)[13] < 0.75;
}

int main() {
	std::ifstream datafile(datafname);
	size_t numInputs;
	std::string line;
	std::vector<std::vector<float>> rawinputs;
	std::vector<float> eventInputs;

	while (std::getline(datafile, line)) {
		eventInputs.clear();
		float val;
		std::stringstream lss(line);
		lss >> val;	//first input assumed to be neuron output
		while (lss >> val)
			eventInputs.push_back(val);
		rawinputs.push_back(eventInputs);
	}

	std::vector<float> mins(rawinputs[0].size());
	std::vector<float> maxes(rawinputs[0].size());
	for (size_t i = 0; i < mins.size(); i++) {
		mins[i] = 99999;
		maxes[i] = -99999;
	}

	for (size_t i = 0; i < rawinputs.size(); i++) {
		for (size_t j = 0; j < rawinputs[i].size(); j++) {
			mins[j] = std::min(mins[j], rawinputs[i][j]);
			maxes[j] = std::max(maxes[j], rawinputs[i][j]);
		}
	}

	std::vector<Histogram2D> hists;
	size_t numBins = 40;
	for (size_t i = 0; i < mins.size(); i++) {
		for (size_t j = i + 1; j < mins.size(); j++) {
			Histogram2D hist;
			hist.initHistogram(mins[i], maxes[i], mins[j], maxes[j], numBins, numBins);
			hists.push_back(hist);
		}
	}
			
	for (size_t i = 0; i < rawinputs.size(); i++) {
		if (!includeEvent(&rawinputs[i]))
			continue;

		size_t histNum = 0;
		for (size_t in1 = 0; in1 < rawinputs[i].size(); in1++) {
			for (size_t in2 = in1 + 1; in2 < rawinputs[i].size(); in2++) {
				hists[histNum].fill(rawinputs[i][in1], rawinputs[i][in2], 1.0f);
				histNum++;
			}
		}
	}

	size_t displayHistNum = 0;
	for (size_t i = 0; i < mins.size(); i++) {
		for (size_t j = i + 1; j < mins.size(); j++) {
			std::stringstream outss;
			outss << outfolder << "disp" << i << "-" << j << "_1";
			std::ofstream outfile(outss.str());
			displayCorrelationHistogram(hists[displayHistNum], &outfile);
			displayHistNum++;
		}
	}
}

float displayCorrelationHistogram(Histogram2D hist, std::ofstream* resfile) {
	float mutInt = 0;
	(*resfile) << "# " << hist.min1 << " " << hist.max1 << " " << hist.min2 << " " << hist.max2 << " " << mutInt << " " << hist.numBins1 << " " << hist.numBins2 << std::endl;
	for (size_t i = 0; i < hist.numBins1; i++) {
		for (size_t j = 0; j < hist.numBins2; j++) {
			size_t pos = i + j*hist.numBins1;
			(*resfile) << hist.bins[pos] << " ";
		}
		(*resfile) << std::endl;
	}
	return mutInt;
}
