
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>

#define datastring ""
//#define datastring "D:/trivialNetworkTest/testdata/"

#define NUM_INPUTS 512
#define NUM_OUTPUTS 4
#define NUM_ADD_FLAGS 1

int main() {
	srand((size_t)time(NULL));

	size_t firstCV = 0;
	std::cout << "Enter first error CV: ";
	std::cin >> firstCV;

	size_t numSets = 0;
	std::cout << "Enter last error CV: ";
	std::cin >> numSets;

	std::vector<float> sample(NUM_INPUTS + NUM_OUTPUTS + NUM_ADD_FLAGS);

	for (size_t cv = firstCV; cv < numSets; cv++) {
		size_t dum = 0;
		size_t numTrainSamples = 0;
		size_t numTestSamples = 0;
		std::stringstream trainss;
		trainss << "fullerrortrainset" << cv + 1;
		FILE* trainfile = fopen(trainss.str().c_str(), "wb");
		fwrite(&dum, sizeof(size_t), 1, trainfile);

		std::stringstream testss;
		testss << "fullerrortestset" << cv + 1;
		FILE* testfile = fopen(testss.str().c_str(), "wb");
		fwrite(&dum, sizeof(size_t), 1, testfile);

		for (size_t s = firstCV; s < numSets; s++) {
			std::stringstream errss;
			errss << "errorset" << s + 1;
			FILE* errfile = fopen(errss.str().c_str(), "rb");
			FILE* outfile = (s == cv ? testfile : trainfile);
			size_t errcount;
			fread(&errcount, sizeof(size_t), 1, errfile);

			if (s == cv)
				numTestSamples += errcount;
			else
				numTrainSamples += errcount;

			for (size_t i = 0; i < errcount; i++) {
				fread(&sample[0], sizeof(float), NUM_INPUTS + NUM_OUTPUTS + NUM_ADD_FLAGS, errfile);
				fwrite(&sample[0], sizeof(float), NUM_INPUTS + NUM_OUTPUTS + NUM_ADD_FLAGS, outfile);
			}
		}

		fseek(trainfile, 0, SEEK_SET);
		fwrite(&numTrainSamples, sizeof(size_t), 1, trainfile);
		fseek(testfile, 0, SEEK_SET);
		fwrite(&numTrainSamples, sizeof(size_t), 1, testfile);
	}
}