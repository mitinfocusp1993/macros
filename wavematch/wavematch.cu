
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <chrono>

struct DataPoint {
	float value;
	int time;
};

std::chrono::system_clock::time_point markedStartTime;
void throwError(std::string err);
void markTime();
long long getTimeSinceMark();

int convertTime(std::string str);
std::string timeToStr(int time);
float getSpline(float left, float right, float leftderiv, float rightderiv, float place);
float evaluateWindow(std::vector<DataPoint>* truthwindow, std::vector<DataPoint>* predictwindow, int offset, float averageWidth, float truthNormalizeFactor, float predictNormalizeFactor);
float getNormalizeFactor(std::vector<DataPoint>* window, float averageWidth);
bool readWindow(std::ifstream* file, size_t start, size_t end, size_t* lineNum, float windowTime, std::vector<DataPoint>* truthwindow, std::vector<DataPoint>* predictwindow, size_t predictcol);

int main() {
	std::string fname;
	std::cout << "Enter name of combined artline-sensor file: ";
	std::cin >> fname;

	size_t predictcol;
	std::cout << "Enter prediction column number: ";
	std::cin >> predictcol;

	float windowTime;
	std::cout << "Enter window length (in s): ";
	std::cin >> windowTime;

	size_t intervalsBetweenSync;
	std::cout << "Enter the number of windows between syncs (use 0 for infinity): ";
	std::cin >> intervalsBetweenSync;

	size_t start;
	size_t end;
	std::cout << "Enter start line: ";
	std::cin >> start;
	std::cout << "Enter end line (0 for whole file): ";
	std::cin >> end;

	float maxOffset;
	float offsetComb;
	std::cout << "Enter max offest (in s): ";
	std::cin >> maxOffset;
	std::cout << "Enter offset comb size (in ms): ";
	std::cin >> offsetComb;

	float avgWidth;
	std::cout << "Enter width of the baseline moving average (in s): ";
	std::cin >> avgWidth;

	int maxOffsetMS = (int)fabs(maxOffset * 1000);
	int offsetCombMS = (int)fabs(offsetComb);

	std::vector<DataPoint> truthwindow;
	std::vector<DataPoint> predictwindow;
	std::ifstream file(fname);
	std::string dum;
	std::getline(file, dum);	//header

	size_t line = 1;
	size_t intNum = 0;
	while (readWindow(&file, start, end, &line, windowTime, &truthwindow, &predictwindow, predictcol)) {
		intNum++;
		float maxR2 = -9999;
		int bestOffset = 0;
		float truthNormalizeFactor = getNormalizeFactor(&truthwindow, avgWidth);
		float predictNormalizeFactor = getNormalizeFactor(&predictwindow, avgWidth);
		for (int off = -maxOffsetMS; off < maxOffsetMS; off += offsetCombMS) {
			float r2 = evaluateWindow(&truthwindow, &predictwindow, off, avgWidth, truthNormalizeFactor, predictNormalizeFactor);
			if (r2 > maxR2) {
				maxR2 = r2;
				bestOffset = off;
			}
		}
		std::cout << "Interval " << intNum << " best offset: " << bestOffset << " at " << maxR2 << " | ";

		for (size_t i = 0; intervalsBetweenSync == 0 || i < intervalsBetweenSync && readWindow(&file, start, end, &line, windowTime, &truthwindow, &predictwindow, predictcol); i++) {
			float r2 = evaluateWindow(&truthwindow, &predictwindow, bestOffset, avgWidth, truthNormalizeFactor, predictNormalizeFactor);
			std::cout << r2 << " ";
		}
		std::cout << std::endl;
	}

	system("pause");
}

void throwError(std::string err) {
	std::cout << err << std::endl;
	throw std::runtime_error(err);
}

void markTime() {
	markedStartTime = std::chrono::high_resolution_clock::now();
}

long long getTimeSinceMark() {
	auto elapsed = std::chrono::high_resolution_clock::now() - markedStartTime;
	long long time = std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
	return time / 1000000;
}

int convertTime(std::string str) {
	int hour = 0;
	int minute = 0;
	int second = 0;
	int msecond = 0;
	char dum;
	std::string pm;

	std::stringstream strs;
	strs << str;
	strs >> hour;
	strs >> dum;	//":"
	strs >> minute;
	strs >> dum;	//":"
	strs >> second;
	strs >> dum;	//"."
	strs >> msecond;

	int time = 3600000 * hour + 60000 * minute + 1000*second + msecond;

	return time;
}
std::string timeToStr(int time) {
	if (time < 0)
		return "0";
	std::stringstream ss;
	ss << time / 3600000 << ":";
	time = time - 3600000 * (time / 3600000);
	ss << time / 60000 << ":";
	time = time - 60000 * (time / 60000);
	ss << time / 1000 << ".";
	time = time - 1000 * (time / 1000);
	return ss.str();
}

bool readWindow(std::ifstream* file, size_t start, size_t end, size_t* lineNum, float windowTime, std::vector<DataPoint>* truthwindow, std::vector<DataPoint>* predictwindow, size_t predictcol) {
	truthwindow->clear();
	predictwindow->clear();

	std::string line;
	while ((*lineNum) < start && std::getline((*file), line)) {
		(*lineNum)++;
	}

	int startTime = 0;
	while ((end == 0 || (*lineNum) <= end)) {
		if (!std::getline((*file), line))
			return false;
		int time;
		std::string dum;

		std::stringstream lss(line);
		std::string tok;
		std::getline(lss, tok, ',');
		time = convertTime(tok);
		if (startTime == 0)
			startTime = time;
		else if (time > startTime + (int)(windowTime * 1000))
			break;

		for (size_t c = 0; c < predictcol; c++)
			std::getline(lss, tok, ',');

		if (tok != "") {
			float predictvalue;
			std::stringstream tss;
			tss.str(tok);
			tss >> predictvalue;
			DataPoint pt;
			pt.value = predictvalue;
			pt.time = time;
			predictwindow->push_back(pt);
		}

		for (size_t c = predictcol; c < 4; c++)
			std::getline(lss, tok, ',');

		std::getline(lss, tok, ',');	//pTime
		std::getline(lss, tok, ',');	//aTime

		std::getline(lss, tok, ',');
		if (tok != "") {
			float truthvalue;
			std::stringstream tss;
			tss.str(tok);
			tss >> truthvalue;
			DataPoint pt;
			pt.value = truthvalue;
			pt.time = time;
			truthwindow->push_back(pt);
		}
		*lineNum++;
	}

	if (end != 0 && *lineNum > end)
		return false;
	return true;
}

float getSpline(float left, float right, float leftderiv, float rightderiv, float place) {
	float a = leftderiv - (right - left);
	float b = -rightderiv + (right - left);
	return (1 - place)*left + place*right + place*(1 - place)*(a*(1 - place) + b*place);
}

float getNormalizeFactor(std::vector<DataPoint>* window, float averageWidth) {
	float maxD = -9999;
	float minD = 9999;

	size_t leftAvgEdge = 0;
	size_t rightAvgEdge = 0;
	float avgTot = 0;
	int msAvgW = (int)fabs(averageWidth * 1000);
	size_t numAvgPts = 0;

	for (size_t i = 0; i < window->size(); i++) {
		int time = (*window)[i].time;
		while (rightAvgEdge < window->size() && (*window)[rightAvgEdge].time <= time + msAvgW) {
			avgTot += (*window)[rightAvgEdge].value;
			numAvgPts++;
			rightAvgEdge++;
		}
		while (leftAvgEdge < window->size() && (*window)[leftAvgEdge].time < time - msAvgW) {
			avgTot -= (*window)[leftAvgEdge].value;
			numAvgPts--;
			leftAvgEdge++;
		}

		float dev = (*window)[i].value - avgTot / numAvgPts;

		if (dev > maxD)
			maxD = dev;
		if (dev < minD)
			minD = dev;
	}

	if (maxD <= minD)
		return 0.0f;
	return 1.0f / (maxD - minD);
}

float evaluateWindow(std::vector<DataPoint>* truthwindow, std::vector<DataPoint>* predictwindow, int offset, float averageWidth, float truthNormalizeFactor, float predictNormalizeFactor) {
	float squaredResidues = 0;
	size_t numMatchedPoints = 0;
	float truthMatchedAvg = 0;
	float truthMatchedSquares = 0;

	size_t nextPredict = 0;

	size_t leftTruthAvgEdge = 0;
	size_t rightTruthAvgEdge = 0;
	float truthAvgTot = 0;
	int msAvgW = (int)fabs(averageWidth * 1000);
	size_t numTruthAvgPts = 0;
	size_t leftPredictAvgEdge = 0;
	size_t rightPredictAvgEdge = 0;
	float predictAvgTot = 0;
	size_t numPredictAvgPts = 0;
	for (size_t i = 0; i < truthwindow->size(); i++) {
		//calculate moving averages
		int time = (*truthwindow)[i].time;
		while (rightTruthAvgEdge < truthwindow->size() && (*truthwindow)[rightTruthAvgEdge].time <= time + msAvgW) {
			truthAvgTot += (*truthwindow)[rightTruthAvgEdge].value;
			numTruthAvgPts++;
			rightTruthAvgEdge++;
		}
		while (leftTruthAvgEdge < truthwindow->size() && (*truthwindow)[leftTruthAvgEdge].time < time - msAvgW) {
			truthAvgTot -= (*truthwindow)[leftTruthAvgEdge].value;
			numTruthAvgPts--;
			leftTruthAvgEdge++;
		}
		while (rightPredictAvgEdge < predictwindow->size() && (*predictwindow)[rightPredictAvgEdge].time + offset <= time + msAvgW) {
			predictAvgTot += (*predictwindow)[rightPredictAvgEdge].value;
			numPredictAvgPts++;
			rightPredictAvgEdge++;
		}
		while (leftPredictAvgEdge < predictwindow->size() && (*predictwindow)[leftPredictAvgEdge].time + offset < time - msAvgW) {
			predictAvgTot -= (*predictwindow)[leftPredictAvgEdge].value;
			numPredictAvgPts--;
			leftPredictAvgEdge++;
		}

		while (nextPredict < predictwindow->size() && (*predictwindow)[nextPredict].time + offset <= time) {
			nextPredict++;
		}
		if (nextPredict >= predictwindow->size())
			break;

		if (nextPredict > 0) {
			float leftderiv;
			float predictTimeWidth = (*predictwindow)[nextPredict].time - (*predictwindow)[nextPredict - 1].time;
			if (nextPredict - 1 == 0)
				leftderiv = ((*predictwindow)[1].value - (*predictwindow)[0].value) / ((*predictwindow)[1].time - (*predictwindow)[0].time) * predictTimeWidth;
			else
				leftderiv = ((*predictwindow)[nextPredict].value - (*predictwindow)[nextPredict - 2].value) / ((*predictwindow)[nextPredict].time - (*predictwindow)[nextPredict - 2].time) * predictTimeWidth;

			float rightderiv;
			if (nextPredict == predictwindow->size() - 1)
				rightderiv = ((*predictwindow)[nextPredict].value - (*predictwindow)[nextPredict - 1].value) / ((*predictwindow)[nextPredict].time - (*predictwindow)[nextPredict - 1].time) * predictTimeWidth;
			else
				rightderiv = ((*predictwindow)[nextPredict + 1].value - (*predictwindow)[nextPredict - 1].value) / ((*predictwindow)[nextPredict + 1].time - (*predictwindow)[nextPredict - 1].time) * predictTimeWidth;

			float place = (time - offset - (*predictwindow)[nextPredict - 1].time) / ((*predictwindow)[nextPredict].time - (*predictwindow)[nextPredict - 1].time);
			float predictvalue = getSpline((*predictwindow)[nextPredict - 1].value, (*predictwindow)[nextPredict].value, leftderiv, rightderiv, place);
			float predictdev = predictNormalizeFactor*(predictvalue - predictAvgTot / numPredictAvgPts);
			float truthdev = truthNormalizeFactor*((*truthwindow)[i].value - truthAvgTot / numTruthAvgPts);

			float unsquare = truthdev - predictdev;
			squaredResidues += unsquare*unsquare;
			numMatchedPoints++;
			truthMatchedAvg += truthdev;
			truthMatchedSquares += truthdev*truthdev;
		}
	}

	squaredResidues /= numMatchedPoints;
	truthMatchedAvg /= numMatchedPoints;
	truthMatchedSquares /= numMatchedPoints;
	float truthVariance = truthMatchedSquares - truthMatchedAvg*truthMatchedAvg;

	return 1.0f - squaredResidues / truthVariance;
}