
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>

#define datastring "D:/momMIMIC/weights/"

int main() {
	std::string infname;
	std::cout << "Enter input file name: ";
	std::cin >> infname;
	infname = datastring + infname;

	std::string outfname;
	std::cout << "Enter output file name: ";
	std::cin >> outfname;
	outfname = datastring + outfname;

	size_t intervalColumn;
	std::cout << "Enter column of interval identification: ";
	std::cin >> intervalColumn;

	size_t numLinesAveraged;
	std::cout << "Enter number of lines to average: ";
	std::cin >> numLinesAveraged;

	std::vector<std::vector<float>> lines;
	std::string line;
	std::ifstream infile(infname);
	std::ofstream outfile(outfname);
	std::string lastInterval = "";

	size_t numLines = 0;
	while (std::getline(infile, line)) {
		std::stringstream lss(line);
		size_t col = 0;
		std::string dum;
		std::vector<float> columns;
		while (lss >> dum) {
			col++;
			if (col == intervalColumn) {
				if (dum != lastInterval) {
					lines.clear();
					numLines = 0;
					lastInterval = dum;
				}
			}
			float val;
			std::stringstream(dum) >> val;
			columns.push_back(val);
		}
		lines.push_back(columns);
		numLines++;

		if (numLines == numLinesAveraged) {
			for (size_t i = 0; i < lines[0].size(); i++) {
				float avg = 0;
				for (size_t j = 0; j < lines.size(); j++) {
					avg += lines[j][i];
				}
				avg /= lines.size();
				outfile << avg << " ";
			}
			outfile << std::endl;

			lines.clear();
			numLines = 0;
		}
	}
}
