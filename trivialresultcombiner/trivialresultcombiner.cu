#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>

#define datastring ""
//#define datastring "D:/trivialNetworkTest/resultCombine/"

#define TEST_COLUMN 6

#define NUM_INPUTS 512
#define NUM_OUTPUTS 4
#define CORRECT_OUTPUT 2
#define NUM_ADD_FLAGS 1
#define NUM_SILENT_OUTPUTS 0
#define HEADER_SIZE 4	//header in binfile created by trivialbinpacker, so it's different from the flag in that program

#define TRAIN_INCLUDESIZE 200000

template <typename T> void randomizeVector(std::vector<T>* vec) {
	for (size_t i = 0; i < vec->size(); i++) {
		size_t j = (RAND_MAX*rand() + rand()) % vec->size();
		T tmp = (*vec)[i];
		(*vec)[i] = (*vec)[j];
		(*vec)[j] = tmp;
	}
}

void main() {
	srand((size_t)time(NULL));

	size_t numCVs;
	std::cout << "Enter number of cv sets: ";
	std::cin >> numCVs;

	size_t numNetworks;
	std::cout << "Enter number of networks: ";
	std::cin >> numNetworks;

	std::string trainfname = "choicetrainset";
	std::string testfname = "choicetestset";

	std::ofstream resfile("resultaverages");

	std::vector<FILE*> choicetrainsets(numCVs);
	std::vector<FILE*> choicetestsets(numCVs);
	size_t dum = 0;
	for (size_t cv = 0; cv < numCVs; cv++) {
		std::stringstream numss;
		numss << "_" << cv + 1;
		choicetrainsets[cv] = fopen((datastring + trainfname + numss.str()).c_str(), "wb");
		fwrite(&dum, sizeof(size_t), 1, choicetrainsets[cv]);
		choicetestsets[cv] = fopen((datastring + testfname + numss.str()).c_str(), "wb");
		fwrite(&dum, sizeof(size_t), 1, choicetestsets[cv]);
	}

	std::vector<size_t> trainCounts(numCVs);
	std::vector<size_t> testCounts(numCVs);
	std::vector<float> networkErrors(numNetworks);
	float averageError = 0;
	float optimalError = 0;
	size_t totalCount = 0;
	size_t numColumns = NUM_INPUTS + NUM_OUTPUTS + NUM_SILENT_OUTPUTS + NUM_ADD_FLAGS;
	for (size_t cv = 0; cv < numCVs; cv++) {
		std::cout << "Starting CV " << cv + 1 << ": ";
		std::vector<std::ifstream> networkfiles(numNetworks);
		std::vector<float> networkOutputs(numNetworks);
		std::vector<float> columns(numColumns);
		std::vector<float> cvNetworkErrors(numNetworks);
		float cvAverageError = 0;
		float cvOptimalError = 0;
		size_t cvCount = 0;

		std::stringstream testss;
		testss << datastring << "testset_" << cv + 1;
		FILE* testset = fopen(testss.str().c_str(), "rb");
		if (testset == NULL) {
			std::cout << "Couldn't find testset_" << cv + 1 << std::endl;
			system("pause");
			return;
		}

		for (size_t n = 0; n < numNetworks; n++) {
			std::stringstream netss;
			netss << datastring << "testresults_c" << cv + 1 << "_n" << n + 1;
			networkfiles[n].open(netss.str());
			if (!networkfiles[n].is_open()) {
				std::cout << "Couldn't find " << netss.str() << std::endl;
				system("pause");
				return;
			}
		}
		size_t numSamples = 0;
		fseek(testset, HEADER_SIZE, SEEK_SET);
		while (fread(&columns[0], sizeof(float), numColumns, testset) == numColumns) {
			numSamples++;
		}

		std::vector<std::vector<size_t>> testsetIndices(numCVs);
		for (size_t testCV = 0; testCV < testsetIndices.size(); testCV++) {
			testsetIndices[testCV].resize(numSamples);
			for (size_t i = 0; i < testsetIndices[testCV].size(); i++)
				testsetIndices[testCV][i] = i;
			randomizeVector(&testsetIndices[testCV]);
		}

		fseek(testset, HEADER_SIZE, SEEK_SET);

		size_t testIndex = 0;
		while (fread(&columns[0], sizeof(float), numColumns, testset) == numColumns) {
			bool testresultDone = false;
			for (size_t n = 0; n < numNetworks; n++) {
				std::string line;
				if (!std::getline(networkfiles[n], line)) {
					testresultDone = true;
					break;
				}
				std::stringstream lss(line);
				for (size_t l = 0; l < NUM_OUTPUTS; l++) {
					float val;
					lss >> val;
					std::stringstream forss;
					forss << columns[l];
					float roundVal;
					forss >> roundVal;
					if (val != roundVal) {
						std::cout << "Unmatched input in testresults_c" << cv + 1 << "_n" << n + 1 << ": " << val << " vs. " << roundVal << std::endl;
						system("pause");
						return;
					}
				}
				for (size_t l = NUM_OUTPUTS; l < TEST_COLUMN - 1; l++) {
					std::string dum;
					lss >> dum;
				}
				lss >> networkOutputs[n];
			}

			if (testresultDone)
				break;

			float corOutput = columns[CORRECT_OUTPUT - 1];
			float minError = 9999;
			float minNetwork = 0;
			float optimalOutput = 0;
			float averageOutput = 0;
			for (size_t n = 0; n < numNetworks; n++) {
				float err = fabs(networkOutputs[n] - corOutput);
				averageOutput += networkOutputs[n];
				if (err < minError) {
					minError = err;
					minNetwork = n;
					optimalOutput = networkOutputs[n];
				}
				cvNetworkErrors[n] += err;
				networkErrors[n] += err;
			}
			averageOutput /= numNetworks;
			cvAverageError += fabs(averageOutput - corOutput);
			averageError += fabs(averageOutput - corOutput);
			cvOptimalError += fabs(optimalOutput - corOutput);
			optimalError += fabs(optimalOutput - corOutput);
			cvCount++;
			totalCount++;

			for (size_t c = 0; c < numCVs; c++) {
				FILE* outfile;
				size_t* count;
				if (c == cv) {
					outfile = choicetestsets[c];
					count = &testCounts[c];
				}
				else {
					outfile = choicetrainsets[c];
					count = &trainCounts[c];
				}
				if (c == cv || testsetIndices[c][testIndex] < TRAIN_INCLUDESIZE) {
					for (size_t n = 0; n < numNetworks; n++) {
						float out;
						if (n == minNetwork)
							out = 1;
						else
							out = -1;
						fwrite(&out, sizeof(float), 1, outfile);
					}

					fwrite(&columns[0], sizeof(float), NUM_OUTPUTS + NUM_SILENT_OUTPUTS + NUM_ADD_FLAGS + NUM_INPUTS, outfile);
					(*count)++;
				}
			}
			testIndex++;
		}

		std::cout << cvCount << " samples read, Net Errors: ";
		resfile << cv + 1 << ": ";
		for (size_t n = 0; n < numNetworks; n++) {
			std::cout << n + 1 << ": " << cvNetworkErrors[n] / cvCount << " ";
			resfile << cvNetworkErrors[n] / cvCount << " ";
		}
		std::cout << "Average: " << cvAverageError / cvCount << " Optimal: " << cvOptimalError / cvCount << std::endl;
		resfile << cvAverageError / cvCount << " " << cvOptimalError / cvCount << std::endl;
	}

	for (size_t cv = 0; cv < numCVs; cv++) {
		fseek(choicetrainsets[cv], 0, SEEK_SET);
		fwrite(&trainCounts[cv], sizeof(size_t), 1, choicetrainsets[cv]);
		fseek(choicetestsets[cv], 0, SEEK_SET);
		fwrite(&testCounts[cv], sizeof(size_t), 1, choicetestsets[cv]);
	}
	std::cout << std::endl << "Total Samples: " << totalCount << " Net Errors: ";
	resfile << "Total: ";
	for (size_t n = 0; n < numNetworks; n++) {
		std::cout << n + 1 << ": " << networkErrors[n] / totalCount << " ";
		resfile << networkErrors[n] / totalCount << " ";
	}
	std::cout << "Average: " << averageError / totalCount << " Optimal: " << optimalError / totalCount << std::endl;
	resfile << averageError / totalCount << " " << optimalError / totalCount << std::endl;
	system("pause");
	system("pause");
}