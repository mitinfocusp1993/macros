
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <cstdio>

#define PI 3.14159265

#define datastring "D:/trivialNetworkTest/"

int main() {
	FILE* trainfile = fopen((datastring + (std::string)"trainset").c_str(), "wb");
	FILE* testfile = fopen((datastring + (std::string)"testset").c_str(), "wb");

	for (size_t i = 0; i < 100000; i++) {
		float in1 = 2.0f*(rand() % 10000)*PI/10000.0f;
		float in2 = 2.0f*(rand() % 10000)*PI/10000.0f;
		float out = sin(in1)*sin(in2);
		fwrite(&out, sizeof(float), 1, trainfile);
		fwrite(&in1, sizeof(float), 1, trainfile);
		fwrite(&in2, sizeof(float), 1, trainfile);
	}
	for (size_t i = 0; i < 100000; i++) {
		float in1 = 2.0f*(rand() % 10000)*PI/10000.0f;
		float in2 = 2.0f*(rand() % 10000)*PI/10000.0f;
		float out = sin(in1)*sin(in2);
		fwrite(&out, sizeof(float), 1, testfile);
		fwrite(&in1, sizeof(float), 1, testfile);
		fwrite(&in2, sizeof(float), 1, testfile);
	}
	fclose(trainfile);
	fclose(testfile);
}
