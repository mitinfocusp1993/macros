
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <random>

#define datastring ""
//#define datastring "D:/trivialNetworkTest/sethsensor/"

#define NUM_INPUTS 512
#define NUM_OUTPUTS 3
#define HEADER_SIZE 8

int main() {
	srand((size_t)time(NULL));

	size_t numFiles = 1;
	std::cout << "Enter number of garbage files to produce: ";
	std::cin >> numFiles;

	size_t numEvents = 2000;
	std::cout << "Enter number of garbage events in each file: ";
	std::cin >> numEvents;

	std::string outfname;
	std::cout << "Enter name of garbage files: ";
	std::cin >> outfname;

	float minOutput = 0;
	float maxOutput = 100;
	std::cout << "Enter minimum output: ";
	std::cin >> minOutput;
	std::cout << "Enter maximum output: ";
	std::cin >> maxOutput;

	std::random_device rd;

	std::mt19937 e2(rd());

	std::normal_distribution<> dist(0, 1);

	for (size_t f = 1; f <= numFiles; f++) {
		std::stringstream outss;
		outss << outfname << "_" << f;
		FILE* outfile = fopen(outss.str().c_str(), "wb");
		char* header[HEADER_SIZE];
		memset(header, 0, HEADER_SIZE);

		fwrite(header, HEADER_SIZE, 1, outfile);

		for (size_t i = 0; i < numEvents; i++) {
			float val = (1.0f*(rand() % 100) / 100.0f)*(maxOutput - minOutput) + minOutput;
			fwrite(&val, sizeof(float), 1, outfile);
			val = 0;
			for (size_t o = 1; o < NUM_OUTPUTS; o++)
				fwrite(&val, sizeof(float), 1, outfile);

			for (size_t in = 0; in < NUM_INPUTS; in++) {
				val = dist(e2);
				fwrite(&val, sizeof(float), 1, outfile);
			}
		}
		fclose(outfile);
	}

	system("pause");
}